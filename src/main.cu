//
// Created by guidocalvano on 1/27/23.
//

#include "../include/main.cuh"
using namespace cv;

using namespace boost::filesystem;


void fileToBytes(std::string inputPathName, uint8_t* byteArray, int offset)
{
  //open file
  std::ifstream infile(inputPathName, std::ifstream::in);
  std::istreambuf_iterator<char> begin{infile}, end;

//get length of file
  uintmax_t size = file_size(inputPathName);
  printf("Size %u: ", size);
  for(int i = 0; i < offset; ++i) begin++;

  std::copy(begin, end, byteArray);
}

void bytesToGreyImageMat(Mat& img, int columns, int rows, uint8_t* imageData)
{
  img.create(columns,rows,CV_8UC1);

  for(int x = 0; x < columns; ++x)
  {
    for(int y = 0; y < rows; ++y) {
      img.at<uint8_t>(y, x) = imageData[y * columns + x];
    }
  }
}


void savePrediction(std::string path, unsigned int imageIndex, Mat img, int label)
{
  std::string filePathName = path +
      std::string("/image_") +
      std::to_string(imageIndex) +
      "_label_" +
      std::to_string(label) +
      ".png";
  imwrite(filePathName, img);
}

void savePredictions(std::string output_path,
                     int imageCount,
                     int rows,
                     int columns,
                     uint8_t* imageData,
                     uint8_t* labelData)
{
  Mat img;

  for(int i = 0; i < imageCount; ++i)
  {
    bytesToGreyImageMat(img, columns, rows, imageData + 28 * 28 * i);
    savePrediction(output_path, i, img, *(labelData + i));
  }
}


#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

class Model
{
  hipdnnHandle_t cudnn;                                    // cudnn interface
  hipblasHandle_t cublas;

  // INPUT
  float *d_input{nullptr};
  float *d_dInput{nullptr};

  // CONVOLUTION LAYER
  // Convolution inputs
  hipdnnTensorDescriptor_t input_descriptor;
  hipdnnTensorDescriptor_t output_descriptor;
  hipdnnFilterDescriptor_t kernel_descriptor;

  // Convolution parameters
  hipdnnConvolutionDescriptor_t convolution_descriptor;

  // Convolution itself
  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  void *d_workspace{nullptr};

  float *d_kernel{nullptr};
  float *d_dKernel{nullptr};

  float *d_convolution_output{nullptr};
  float *d_dConvolution_output{nullptr};
  size_t workspace_bytes;
  // DENSE LAYER
  float* d_denseWeights;
  float *d_dDenseWeights{nullptr};

  // ACTIVATION FUNCTION
  float *d_activation;
  float *d_dActivation;
  hipdnnActivationDescriptor_t activation_descriptor;

  float *d_output{nullptr};
  float *d_dOutput{nullptr};

  float* h_output;

  int columnCount;
  int rowCount;
  int colorCount;
  int labelClasses;

  public:
  Model(int columnCount, int rowCount, int colorCount, int labelClasses) {
    this->columnCount = columnCount;
    this->rowCount = rowCount;
    this->colorCount = colorCount;
    this->labelClasses = labelClasses;
    // input
    hipdnnCreate(&cudnn);
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/1,
        /*channels=*/colorCount,
        /*image_height=*/rowCount,
        /*image_width=*/columnCount));

    // output
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
        /*format=*/HIPDNN_TENSOR_NHWC,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*batch_size=*/1,
        /*channels=*/colorCount,
        /*image_height=*/rowCount,
        /*image_width=*/columnCount));

    // kernel
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
        /*dataType=*/HIPDNN_DATA_FLOAT,
        /*format=*/HIPDNN_TENSOR_NCHW,
        /*out_channels=*/colorCount,
        /*in_channels=*/colorCount,
        /*kernel_height=*/3,
        /*kernel_width=*/3));
    // DESCRIBE CONVOLUTION
    // convolution descriptor

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
        /*pad_height=*/1,
        /*pad_width=*/1,
        /*vertical_stride=*/1,
        /*horizontal_stride=*/1,
        /*dilation_height=*/1,
        /*dilation_width=*/1,
        /*mode=*/HIPDNN_CROSS_CORRELATION,
        /*computeType=*/HIPDNN_DATA_FLOAT));

    checkCUDNN(
        hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            kernel_descriptor,
                                            convolution_descriptor,
                                            output_descriptor,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
            /*memoryLimitInBytes=*/0,
                                            &convolution_algorithm));

    // DEFINE MEMORY REQUIREMENTS
    workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       input_descriptor,
                                                       kernel_descriptor,
                                                       convolution_descriptor,
                                                       output_descriptor,
                                                       convolution_algorithm,
                                                       &workspace_bytes));
    std::cerr << "Workspace size: " << (workspace_bytes) << "B"
              << std::endl;

    hipMalloc(&d_workspace, workspace_bytes);

    int image_bytes = colorCount * rowCount * columnCount * sizeof(float);

    hipMalloc(&d_input, image_bytes);
    hipMalloc(&d_dInput, image_bytes);

    hipblasStatus_t status;

    status=cublasAlloc(rowCount*columnCount*colorCount, sizeof(float), (void**)&d_convolution_output);
    status=cublasAlloc(rowCount*columnCount*colorCount, sizeof(float), (void**)&d_dConvolution_output);

    status=cublasAlloc(rowCount*columnCount*colorCount * labelClasses,sizeof(float),(void**)&d_denseWeights);
    status=cublasAlloc(rowCount*columnCount*colorCount * labelClasses,sizeof(float),(void**)&d_dDenseWeights);

    status=cublasAlloc(labelClasses,sizeof(float),(void**)&d_activation);
    status=cublasAlloc(labelClasses,sizeof(float),(void**)&d_dActivation);

    hipMalloc(&d_output, labelClasses * sizeof (float));
    hipMalloc(&d_dOutput, labelClasses * sizeof (float));

    initKernelWeightsRandomUniform(colorCount, colorCount);
    initDenseWeightsRandom();
    // dense layer is matrix multiplaction
    // https://docs.nvidia.com/deeplearning/performance/dl-performance-fully-connected/index.html
    hipblasCreate(&cublas);

    // ACTIVATION FUNCTION
    checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
    checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
        /*mode=*/HIPDNN_ACTIVATION_SIGMOID,
        /*reluNanOpt=*/HIPDNN_PROPAGATE_NAN,
        /*relu_coef=*/0));
  }

  void initWeightsEdgeDetector(int kernelCount, int channelCount)
  {
    float h_kernel[kernelCount][channelCount][3][3];

    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMalloc(&d_dKernel, sizeof(h_kernel));

    // Mystery kernel
    const float kernel_template[3][3] = {
        {1, 1, 1},
        {1, -8, 1},
        {1, 1, 1}
    };

    for (int kernel = 0; kernel < kernelCount; ++kernel) {
      for (int channel = 0; channel < channelCount; ++channel) {
        for (int row = 0; row < 3; ++row) {
          for (int column = 0; column < 3; ++column) {
            h_kernel[kernel][channel][row][column] = kernel_template[row][column];
          }
        }
      }
    }
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
  }

  void initKernelWeightsRandomUniform(int kernelCount, int channelCount)
  {
    float h_kernel[kernelCount][channelCount][3][3];

    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMalloc(&d_dKernel, sizeof(h_kernel));

    for (int kernel = 0; kernel < kernelCount; ++kernel) {
      for (int channel = 0; channel < channelCount; ++channel) {
        for (int row = 0; row < 3; ++row) {
          for (int column = 0; column < 3; ++column) {
            float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            h_kernel[kernel][channel][row][column] = r * 2.0 - 1.0;
          }
        }
      }
    }
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
  }

  void initDenseWeightsRandom()
  {
    int valueCount = rowCount * columnCount * colorCount * labelClasses;
    float* h_value = new float[valueCount];

    for(int v = 0; v < valueCount; ++v)
    {
      float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

      h_value[v] = r * 2.0 - 1.0;
      // printf("%.2f ", h_value[v]);
    }
    hipMemcpy(d_denseWeights, h_value, valueCount * sizeof(float), hipMemcpyHostToDevice);
  }

  void printNetwork()
  {
    printKernel();
    printDense();
  }

  void printKernel()
  {
    float h_kernel[colorCount][colorCount][3][3];
    hipMemcpy(h_kernel, d_kernel, sizeof(h_kernel), hipMemcpyDeviceToHost);

    for (int kernel = 0; kernel < colorCount; ++kernel) {
      for (int channel = 0; channel < colorCount; ++channel) {
        for (int row = 0; row < 3; ++row) {
          for (int column = 0; column < 3; ++column) {
            printf("%.3f ", h_kernel[kernel][channel][row][column]);
          }
          printf("\n");
        }
      }
    }
  }

  void printDense()
  {
    int valueCount = rowCount * columnCount * colorCount * labelClasses;
    float* h_value = new float[valueCount];

    hipMemcpy(h_value, d_denseWeights, valueCount * sizeof(float), hipMemcpyDeviceToHost);

    for(int v = 0; v < valueCount; ++v)
      printf("%.2f ", h_value[v]);
//    for(int l = 0; l < labelClasses; ++l)
//    {
//      for(int r = 0; r < rowCount; ++r)
//      for(int c = 0; c < columnCount; ++c)
//      for(int k = 0; k < colorCount; ++k)
//        printf("%.2f ", h_value[c * rowCount * colorCount * labelClasses + r * colorCount * labelClasses + k * labelClasses + l]);
//      printf("\n");
//    }
  }

  void deleteModel()
  {
    delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    cublasFree(d_convolution_output);
    cublasFree(d_dConvolution_output);

    cublasFree(d_denseWeights);
    cublasFree(d_dDenseWeights);

    hipdnnDestroyActivationDescriptor(activation_descriptor);

    hipblasDestroy(cublas);
    hipdnnDestroy(cudnn);
  }

  void backwardPass()
  {
    const float alpha = 1, beta = 0;

    // activation backprop
    hipdnnStatus_t status = hipdnnActivationBackward(
                                         cudnn,
                                         activation_descriptor,
                                         &alpha,
                                         output_descriptor,
                                         d_output,
                                         output_descriptor,
                                         d_dOutput,
                                         output_descriptor,
                                         d_activation  ,
                                         &beta,
                                         output_descriptor,
                                         d_dActivation);
    // backpropagate to previous layer
    // C = alpha * op(A) * op(B) + beta * C
    hipblasSgemm('n', // op(A) = A
                't', // op(B) = transpose(B)
                1, //rows in C and A
                columnCount * rowCount * colorCount, // Columns in C and B
                labelClasses, // columns in A and rows in B, i.e. the dimensions that are dot producted away
                1,
                d_dActivation,
                1,
                d_denseWeights,
                columnCount * rowCount * colorCount,
                0,
                d_dConvolution_output,
                1);

    // backpropagate to weights

    hipblasSgemm('n',
                'n',
                columnCount * rowCount * colorCount,
                labelClasses,
                1,
                1,
                d_convolution_output,
                columnCount * rowCount * colorCount,
                d_dActivation,
                1,
                0,
                d_dDenseWeights,
                columnCount * rowCount * colorCount);
    status = hipdnnConvolutionBackwardFilter(
                                            cudnn,
                                             &alpha,
                                            input_descriptor,
                                         d_input,
                                         output_descriptor,
                                         d_dConvolution_output,
                                convolution_descriptor,
                                            HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1,
                                            d_workspace,
                                            workspace_bytes,
                                      &beta,
               kernel_descriptor,
                                            d_dKernel);

    status =  hipdnnConvolutionBackwardData(
                                               cudnn,
                                               &alpha,
                                               kernel_descriptor,
                                               d_kernel,
                                               output_descriptor,
                                               d_dConvolution_output,
                                               convolution_descriptor,
                                               HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,
                                               d_workspace,
                                               workspace_bytes,
                                               &beta,
                                               input_descriptor,
                                               d_dInput);



  }

  void forwardPass()
  {

    // forward pass through convolution
    // hipMemcpy(d_input, imageData, rowCount * columnCount * colorCount * sizeof(float), hipMemcpyHostToDevice);
//    hipMemset(d_output, 0, image_bytes);

    // DEBUG
    float* h_input = new float[rowCount * columnCount * colorCount * sizeof(float)];

    hipMemcpy(h_input, d_input, rowCount * columnCount * colorCount * sizeof(float), hipMemcpyDeviceToHost);

    // for(int v = 0; v < rowCount * columnCount * colorCount; ++v) printf("%.2f ", h_input[v]);
    // printf("\n");
    // END DEBUG

    const float alpha = 1, beta = 0;
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       d_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       d_convolution_output));

    // DEBUG
    float* h_convolution_output = new float[rowCount * columnCount * colorCount * sizeof(float)];

    hipMemcpy(h_convolution_output, d_convolution_output, rowCount * columnCount * colorCount * sizeof(float), hipMemcpyDeviceToHost);

    // for(int v = 0; v < rowCount * columnCount * colorCount; ++v) printf("%.2f ", h_convolution_output[v]);
    // printf("\n");
    // END DEBUG


    // dense layer is matrix multiplication.
    // https://docs.nvidia.com/deeplearning/performance/dl-performance-fully-connected/index.htmls
    hipblasSgemm('n','n',1,labelClasses, columnCount * rowCount * colorCount,1,d_convolution_output,1,d_denseWeights,columnCount * rowCount * colorCount,0,d_activation,1);

    // DEBUG
    float* h_activation = new float[labelClasses* sizeof(float)];

    hipMemcpy(h_activation, d_activation, labelClasses * sizeof(float), hipMemcpyDeviceToHost);

    // for(int v = 0; v < labelClasses; ++v) printf("%.2f ", h_activation[v]);
    // END DEBUG
    // Perform the forward pass of the activation
    checkCUDNN(hipdnnActivationForward(cudnn,
                                      activation_descriptor,
                                      &alpha,
                                      output_descriptor,
                                      d_activation,
                                      &beta,
                                      output_descriptor,
                                      d_output));


    // COPY MEMORY TO HOST
    h_output = new float[labelClasses* sizeof(float)];
    hipMemcpy(h_output, d_output, labelClasses * sizeof(float), hipMemcpyDeviceToHost);

    // DEBUG
    // for(int v = 0; v < labelClasses; ++v) printf("%.2f ", h_output[v]);
    // free memory and copy info back to host

  }

  void computeLoss(int correctPrediction)
  {
    float h_dOutput[labelClasses];

    for(int l = 0; l < labelClasses; l++)
    {
      h_dOutput[l] = -h_output[l];

      if(l == correctPrediction)
      {
        h_dOutput[l] = 1.0 - h_output[l];
      }
    }
//    printf("loss %.6f %.6f %.6f %.6f %.6f %.6f %.6f %.6f %.6f %.6f",
//           h_dOutput[0],
//           h_dOutput[1],
//           h_dOutput[2],
//           h_dOutput[3],
//           h_dOutput[4],
//           h_dOutput[5],
//           h_dOutput[6],
//           h_dOutput[7],
//           h_dOutput[8],
//           h_dOutput[9]
//           );
    hipMemcpy(d_dOutput, h_dOutput, labelClasses * sizeof(float), hipMemcpyHostToDevice);
  }

  int outputToPrediction()
  {
    float highestProbability = -1.0;
    int highestLabel = -1;

    for(int l = 0; l < labelClasses; l++)
    {
      float probability = h_output[l];

      if(probability > highestProbability)
      {
        highestProbability = probability;
        highestLabel = l;
      }
    }
    return highestLabel;
  }

  void updateWeights(float learningRate)
  {
    updateKernelWeights(learningRate, colorCount, colorCount);
    updateDenseWeights(learningRate);
  }

  void updateKernelWeights(float learningRate, int kernelCount, int colorCount)
  {
    int valueCount = 3 * 3 * kernelCount * colorCount;

    hipblasSaxpy(valueCount, learningRate, d_dKernel, 1, d_kernel, 1);
  }

  void updateDenseWeights(float learningRate)
  {
    int valueCount = rowCount * columnCount * colorCount * labelClasses;

    hipblasSaxpy(valueCount, learningRate, d_dDenseWeights, 1, d_denseWeights, 1);
  }

  uint8_t trainStep(uint8_t* imageData, uint8_t label)
  {
    set_image_float(imageData);

    forwardPass();

    computeLoss(label);

    backwardPass();
    updateWeights(.1);

    return outputToPrediction();
  }

  uint8_t predictStep(uint8_t* imageData)
  {
    set_image_float(imageData);

    forwardPass();

    uint8_t prediction = outputToPrediction();
    return prediction;
  }

  void set_image_float(uint8_t* imageData)
  {
    // used to cast to float
    float* h_imageData = (float*) malloc(rowCount * columnCount * colorCount * sizeof(float));

    for(int x =0; x < columnCount; ++x)
      for(int y=0; y < rowCount; ++y)
        for(int c=0; c < colorCount; ++c)
          h_imageData[
              y * columnCount * colorCount + x * colorCount + c] =
              ((float) imageData[
                  y * columnCount * colorCount + x * colorCount + c]) / 255.0;


    hipMemcpy((void*) d_input, (void*) h_imageData, rowCount*colorCount*columnCount * sizeof(float), hipMemcpyHostToDevice);

    free(h_imageData);
    return;
  }

  int imageSize()
  {
    return rowCount * columnCount * colorCount;
  }

  void trainEpoch(uint8_t* imageData, int imageCount, uint8_t* correct, int epochCount)
  {
    for(int i = 0; i < imageCount; ++i)
      trainStep(imageData + imageSize() * i, correct[i]);
  }

  void trainEpochs(uint8_t* imageData, int imageCount, uint8_t* correct, int epochCount)
  {
    for(int n=0; n < epochCount; ++n)
      trainEpoch(imageData, imageCount, correct, epochCount);
  }

  void predictAll(uint8_t* imageData, int imageCount, uint8_t* results)
  {
    for(int i = 0; i < imageCount; ++i)
      results[i] = predictStep(imageData + imageSize() * i);
  }

  float computeAccuracy(uint8_t* imageData, int imageCount, uint8_t* correct)
  {
    uint8_t* predictions = new uint8_t[imageCount];

    predictAll(imageData, imageCount, predictions);

    int correctCount = 0;

    for(int i = 0; i < imageCount; ++i)
      correctCount += predictions[i] == correct[i];

    return ((float) correctCount) / ((float) imageCount);
  }
};

void run(int imageCount, int columnCount, int rowCount, int colorCount, int labelClasses, uint8_t* imageData, uint8_t* labelData, uint8_t* predictionData)
{
  Model model(columnCount, rowCount, colorCount, labelClasses);

  std::cout << "output " << (int) model.predictStep(imageData) << std::endl;

  // return;
  for(int m = 0; m < 11; ++m)
  {
    model.trainEpoch(imageData, imageCount, labelData, 1);
    float accuracy = model.computeAccuracy(imageData, imageCount, labelData);
    printf("Accuracy %.6f\n", accuracy);
  }


  return;
  float* h_imageData = (float*) malloc(imageCount * rowCount * columnCount * colorCount * sizeof(float));

  for(int i = 0; i < imageCount; ++i)
    for(int x =0; x < columnCount; ++x)
      for(int y=0; y < rowCount; ++y)
        for(int c=0; c < colorCount; ++c)
          h_imageData[i * columnCount * rowCount * colorCount +
                  y * columnCount * colorCount + x * colorCount + c] =
                      (float) imageData[i * columnCount * rowCount * colorCount +
                      y * columnCount * colorCount + x * colorCount + c];

  float* d_imageData{nullptr};
  hipMalloc(&d_imageData, sizeof(h_imageData));
  hipMemcpy((void*) d_imageData, (void*) h_imageData, imageCount*rowCount*colorCount*columnCount * sizeof(float), hipMemcpyHostToDevice);


  model.forwardPass();

  int prediction = model.outputToPrediction();

  model.computeLoss(labelData[0]);

  model.backwardPass();
  model.updateWeights(.1);

  return;

}

void loadTrainingData(uint8_t* imageData, uint8_t* labelData)
{
//   std::vector<uint8_t> imageByteVector;
//   std::vector<uint8_t> labelByteVector;
  int imageOffset = 4 * 4;
  int labelOffset = 2 * 4;
   fileToBytes("./data/input/training_image.idx3", imageData, imageOffset);
   fileToBytes("./data/input/training_label.idx1", labelData, labelOffset);

//
//   std::cout << " image byte vector size "<< imageByteVector.size() - imageOffset << std::endl;
//   std::cout << " label byte vector size "<< labelByteVector.size() - labelOffset << std::endl;
//
//   std::memcmp(imageData, imageByteVector.data() + imageOffset, imageByteVector.size() - imageOffset);
//   std::memcmp(labelData, labelByteVector.data() + labelOffset, labelByteVector.size() - labelOffset);
}


int main(int argc, char *argv[])
{
  if(argc < 3) {
      printf("Syntax: main.exe inputpath outputpath");
      return 1;
  }
  hipDeviceReset();
  // hipError_t err = hipDeviceReset();
  // hardcoded image data because c++ is acting difficult
  int32_t imageCount = 60000;
  int32_t columns = 28;
  int32_t rows = 28;
  int32_t labelClasses = 10;
  int colorCount = 1;

  // allocating this as arrays on the stack causes a segfault, because apparently the stack doesn't like it if you
  // allocate 47MB on it...
  uint8_t* imageData = new uint8_t[rows*columns*imageCount*colorCount];
  uint8_t* labelData = new uint8_t[imageCount];
  uint8_t* predictionData = new uint8_t[imageCount];

  // this will hideously crash if you don't have the right dimensions but my goal is to just explore cudnn so I don't care
  loadTrainingData(imageData, labelData);

  run(imageCount, columns, rows, colorCount, labelClasses, imageData, labelData, predictionData);

  savePredictions(std::string("./data/output"), imageCount, rows, columns, imageData, labelData);//, labelClasses);
  // err = hipDeviceReset();
  delete imageData;
  delete labelData;
  delete predictionData;
}